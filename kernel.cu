#include "hip/hip_runtime.h"
#include <math_functions.h>
#include "pso_cluster.h"

/*
 * Get euclidean distance between 2 pixels
 */
__host__ __device__ 
float devGetDistance(int *first, int *second, int channel)
{
    float total = 0.0f;

    for (int i = 0; i < channel; i++)
    {
        int res = (first[i] - second[i]);
        total += res * res;
    }

    return sqrt(total);
}

/*
 * Get error for given centroids
 */
__host__ __device__ 
float devFitness(short* assignMat, int* datas, int* centroids, int data_size, 
                 int cluster_size, int channel)
{
    float total = 0.0f;

    for (int i = 0; i < cluster_size; i++)
    {
        float subtotal = 0.0f;

        for (int j = 0; j < data_size; j++)
        {
            if (assignMat[j] == i)
                subtotal += devGetDistance(&datas[j * channel],
                                           &centroids[i * channel], channel);
        }

        total += subtotal / data_size;
    }

    return total / cluster_size;
}

/*
 * Assign pixels to centroids
 */
__host__ __device__ 
void devAssignDataToCentroid(short *assignMat, int *datas, int *centroids, 
                             int data_size, int cluster_size, int channel)
{
    for (int i = 0; i < data_size; i++)
    {
        int nearestCentroidIdx = 0;
        float nearestCentroidDist = INF;

        for (int j = 0; j < cluster_size; j++)
        {
            float nearestDist = devGetDistance(&datas[i * channel],
                                               &centroids[j * channel], channel);

            if (nearestDist < nearestCentroidDist)
            {
                nearestCentroidDist = nearestDist;
                nearestCentroidIdx = j;
            }
        }

        assignMat[i] = nearestCentroidIdx;
    }
}

/*
 * Initialize necessary variables for PSO
 */
void initialize(int *positions, int *velocities, int *pBests, int *gBest, 
                const Data* datas, int data_size, int particle_size,
                int cluster_size, int channel)
{
    for (int i = 0; i < particle_size * cluster_size * channel; i+= channel)
    {
        int rand = round(getRandom(0, data_size - 1));

        for(int j = 0; j < channel; j++)
        {
            positions[i + j] = datas[rand].info[j];
            pBests[i + j] = datas[rand].info[j];
            velocities[i + j] = 0;
        }
    }

    for(int i = 0; i < cluster_size * channel; i++)
        gBest[i] = pBests[i];
}

/*
 * Kernel to update particle
 */
__global__ void kernelUpdateParticle(int *positions, int *velocities, 
                                     int *pBests, int *gBest, short *posAssign, 
                                     int* datas, float rp, float rg, 
                                     int data_size, int particle_size, 
                                     int cluster_size, int channel)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= particle_size * cluster_size * channel)
        return;

    // Update particle velocity and position
    velocities[i] = (int)lroundf(OMEGA * velocities[i]
            + c1 * rp * (pBests[i] - positions[i])
            + c2 * rg * (gBest[i % (cluster_size * channel)] - positions[i]));

    positions[i] += velocities[i];
}

/*
 * Kernel to update particle
 */
__global__ void kernelUpdatePBest(int *positions, int *pBests, short *posAssign, 
                                  short *pBestAssign, int* datas, int data_size,
                                  int particle_size, int cluster_size,
                                  int channel)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetParticle = i * cluster_size * channel;
    int offsetAssign = i * data_size;

    if(i >= particle_size)
        return;

    devAssignDataToCentroid(&posAssign[offsetAssign], datas, 
                            &positions[offsetParticle], data_size, 
                            cluster_size, channel);

    // Update pBest
    if (devFitness(&posAssign[offsetAssign], datas, &positions[offsetParticle], 
                   data_size, cluster_size, channel)
            < devFitness(&pBestAssign[offsetAssign], datas, 
                         &pBests[offsetParticle], data_size, cluster_size,
                         channel))
    {
        // Update pBest position
        for (int k = 0; k < cluster_size * channel; k++)
            pBests[offsetParticle + k] = positions[offsetParticle + k];

        // Update pBest assignment matrix
        for(int k = 0; k < data_size; k++)
            pBestAssign[offsetAssign + k] = posAssign[offsetAssign + k];
    }
}

/*
 * Wrapper to initialize and running PSO on device
 */
GBest devicePsoClustering(Data *datas, int *flatDatas, int data_size,
                          int channel, int particle_size, int cluster_size,
                          int max_iter)
{
    // Initialize host memory
    int *positions = new int[particle_size * cluster_size * channel];
    int *velocities = new int[particle_size * cluster_size * channel];
    short *posAssign = new short[particle_size * data_size];
    short *gBestAssign = new short[data_size];  

    size_t size = sizeof(int) * particle_size * cluster_size * channel;
    size_t assign_size = sizeof(short) * particle_size * data_size;
        
    int *pBests;
    int *gBest;
    short *pBestAssign;

    hipHostAlloc((void**)&pBests, size, hipHostMallocDefault);
    hipHostAlloc((void**)&gBest, sizeof(int) * cluster_size * channel,
                  hipHostMallocDefault);
    hipHostAlloc((void**)&pBestAssign, assign_size, hipHostMallocDefault);

    // Initialize assignment matrix to cluster 0
    for(int i = 0; i < particle_size * data_size; i++)
    {
        posAssign[i] = 0;
        pBestAssign[i] = 0;

        if(i < data_size)
            gBestAssign[i] = 0;
    }

    initialize(positions, velocities, pBests, gBest, datas, data_size, 
               particle_size, cluster_size, channel);

    // Initialize device memory
    int *devPositions, *devVelocities, *devPBests, *devGBest;
    short *devPosAssign, *devPBestAssign;
    int *devDatas;

    hipMalloc((void**)&devPositions, size);
    hipMalloc((void**)&devVelocities, size);
    hipMalloc((void**)&devPBests, size);
    hipMalloc((void**)&devGBest, sizeof(int) * cluster_size * channel);
    hipMalloc((void**)&devPosAssign, assign_size);
    hipMalloc((void**)&devPBestAssign, assign_size);
    hipMalloc((void**)&devDatas, sizeof(int) * data_size * channel);

    // Copy Data from host to device
    hipMemcpy(devPositions, positions, size, hipMemcpyHostToDevice);
    hipMemcpy(devVelocities, velocities, size, hipMemcpyHostToDevice);
    hipMemcpy(devPBests, pBests, size, hipMemcpyHostToDevice);
    hipMemcpy(devGBest, gBest, sizeof(int) * cluster_size * channel,
               hipMemcpyHostToDevice);
    hipMemcpy(devPosAssign, posAssign, assign_size, hipMemcpyHostToDevice);
    hipMemcpy(devPBestAssign, pBestAssign, assign_size, 
               hipMemcpyHostToDevice);
    hipMemcpy(devDatas, flatDatas, sizeof(int) * data_size * channel,
               hipMemcpyHostToDevice);

    // Threads and blocks number
    int threads = 32;
    int blocksPart = (particle_size / threads) + 1;
    int blocksFull = (particle_size * cluster_size * channel / threads) + 1;

    // Create event for asynchronous processing between memcpy and kernel
    hipStream_t stream0, stream1;

    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    // Run kernel for the first time to populate memory
    kernelUpdateParticle<<<blocksFull, threads>>>
        (devPositions, devVelocities, devPBests, devGBest, devPosAssign, 
         devDatas, getRandomClamped(), getRandomClamped(), data_size, 
         particle_size, cluster_size, channel);

    kernelUpdatePBest<<<blocksPart, threads>>>
        (devPositions, devPBests, devPosAssign, devPBestAssign, devDatas, 
         data_size, particle_size, cluster_size, channel);

    hipDeviceSynchronize();

    // Iteration
    for (int iter = 0; iter < max_iter; iter++)
    {
        float rp = getRandomClamped();
        float rg = getRandomClamped();

        // Copy result of last iteration to host memory, async using stream-0
        hipMemcpyAsync(pBests, devPBests, size, hipMemcpyDeviceToHost, 
                        stream0);
        hipMemcpyAsync(pBestAssign, devPBestAssign, assign_size, 
                        hipMemcpyDeviceToHost, stream0);
        
        // Run kernel, async using stream-1
        kernelUpdateParticle<<<blocksFull, threads, 0, stream1>>>
            (devPositions, devVelocities, devPBests, devGBest, devPosAssign, 
             devDatas, rp, rg, data_size, particle_size, cluster_size, channel);

        kernelUpdatePBest<<<blocksPart, threads, 0, stream1>>>
            (devPositions, devPBests, devPosAssign, devPBestAssign, devDatas, 
             data_size, particle_size, cluster_size, channel);

        // Calculate gBest async between GPU and CPU
        for(int i = 0; i < particle_size; i++)
        {
            // Get slice of array
            int offsetParticle = i * cluster_size * channel;
            int offsetAssign = i * data_size;

            // Compare pBest and gBest
            if (devFitness(&pBestAssign[offsetAssign], flatDatas, 
                           &pBests[offsetParticle], data_size, cluster_size,
                           channel)
                < devFitness(gBestAssign, flatDatas, gBest, data_size, 
                             cluster_size, channel))
            {
                // Update gBest position
                for (int k = 0; k < cluster_size * channel; k++)
                    gBest[k] = pBests[offsetParticle + k];

                // Update gBest assignment matrix
                for(int k = 0; k < data_size; k++)
                    gBestAssign[k] = pBestAssign[offsetAssign + k];
            }
        }

        // Copy gBest Data back to GPU
        hipMemcpyAsync(devGBest, gBest, sizeof(int) * cluster_size * channel,
                        hipMemcpyHostToDevice, stream0);
    }

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);

    // Copy gBest from device to host
    hipMemcpy(gBest, devGBest, sizeof(int) * cluster_size * channel,
               hipMemcpyDeviceToHost);

    // Cleanup
    delete[] positions;
    delete[] velocities;
    delete[] posAssign;

    hipFree(devPositions);
    hipFree(devVelocities);
    hipFree(devDatas); 
    hipFree(devPosAssign);
    hipFree(devPBests);
    hipFree(devGBest);
    hipFree(devPBestAssign);

    hipHostFree(pBests);
    hipHostFree(pBestAssign);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

    GBest gBestReturn;
    gBestReturn.gBestAssign = gBestAssign;
    gBestReturn.arrCentroids = gBest;
    gBestReturn.quantError = devFitness(gBestReturn.gBestAssign, flatDatas,
                                        gBestReturn.arrCentroids, data_size,
                                        cluster_size, channel);

    return gBestReturn;
}
