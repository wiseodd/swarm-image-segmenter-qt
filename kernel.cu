#include "hip/hip_runtime.h"
#include <math_functions.h>
#include <thrust/extrema.h>
#include "pso_cluster.h"

/*
 * Get euclidean distance between 2 pixels
 */
__host__ __device__
float devGetDistance(int *first, int *second, int channel)
{
    float total = 0.0f;

    for (int i = 0; i < channel; i++)
    {
        int res = (first[i] - second[i]);
        total += res * res;
    }

    return sqrt(total);
}

/*
 * Get error for given centroids
 */
__host__ __device__
float devFitness(short* assignMat, int* datas, int* centroids, int data_size,
                 int cluster_size, int channel)
{
    float total = 0.0f;

    for (int i = 0; i < cluster_size; i++)
    {
        float subtotal = 0.0f;

        for (int j = 0; j < data_size; j++)
        {
            if (assignMat[j] == i)
                subtotal += devGetDistance(&datas[j * channel],
                                           &centroids[i * channel], channel);
        }

        total += subtotal / data_size;
    }

    return total / cluster_size;
}

/*
 * Assign pixels to centroids
 */
__host__ __device__
void devAssignDataToCentroid(short *assignMat, int *datas, int *centroids,
                             int data_size, int cluster_size, int channel)
{
    for (int i = 0; i < data_size; i++)
    {
        int nearestCentroidIdx = 0;
        float nearestCentroidDist = INF;

        for (int j = 0; j < cluster_size; j++)
        {
            float nearestDist = devGetDistance(&datas[i * channel],
                                               &centroids[j * channel], channel);

            if (nearestDist < nearestCentroidDist)
            {
                nearestCentroidDist = nearestDist;
                nearestCentroidIdx = j;
            }
        }

        assignMat[i] = nearestCentroidIdx;
    }
}

/*
 * Initialize necessary variables for PSO
 */
void initialize(int *positions, int *velocities, int *pBests, int *gBest,
                const Data* datas, int data_size, int particle_size,
                int cluster_size, int channel)
{
    for (int i = 0; i < particle_size * cluster_size * channel; i+= channel)
    {
        int rand = round(getRandom(0, data_size - 1));

        for(int j = 0; j < channel; j++)
        {
            positions[i + j] = datas[rand].info[j];
            pBests[i + j] = datas[rand].info[j];
            velocities[i + j] = 0;
        }
    }

    for(int i = 0; i < cluster_size * channel; i++)
        gBest[i] = pBests[i];
}

/*
 * Kernel to update particle
 */
__global__ void kernelUpdateParticle(int *positions, int *velocities,
                                     int *pBests, int *gBest, short *posAssign,
                                     int* datas, float rp, float rg,
                                     int data_size, int particle_size,
                                     int cluster_size, int channel)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= particle_size * cluster_size * channel)
        return;

    // Update particle velocity and position
    velocities[i] = (int)lroundf(OMEGA * velocities[i]
                    + c1 * rp * (pBests[i] - positions[i])
                    + c2 * rg * (gBest[i % (cluster_size * channel)]
                    - positions[i]));

    positions[i] += velocities[i];
}

/*
 * Kernel to update particle
 */
__global__ void kernelUpdatePBest(int *positions, int *pBests, short *posAssign,
                                  short *pBestAssign, float *fitnesses,
                                  int* datas, int data_size, int particle_size,
                                  int cluster_size, int channel)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetParticle = i * cluster_size * channel;
    int offsetAssign = i * data_size;

    if(i >= particle_size)
        return;

    devAssignDataToCentroid(&posAssign[offsetAssign], datas,
                            &positions[offsetParticle], data_size, cluster_size,
                            channel);

    fitnesses[i] = devFitness(&pBestAssign[offsetAssign], datas,
                              &pBests[offsetParticle], data_size, cluster_size,
                              channel);

    // Update pBest
    if (devFitness(&posAssign[offsetAssign], datas, &positions[offsetParticle],
                   data_size, cluster_size, channel)
        < fitnesses[i])
    {
        // Update pBest position
        for (int k = 0; k < cluster_size * channel; k++)
            pBests[offsetParticle + k] = positions[offsetParticle + k];

        // Update pBest assignment matrix
        for(int k = 0; k < data_size; k++)
            pBestAssign[offsetAssign + k] = posAssign[offsetAssign + k];
    }
}

__global__ void kernelUpdateGBest(int *gBest, int *pBests, int offset,
                                  int cluster_size, int channel)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= cluster_size * channel)
        return;

    gBest[i] = pBests[offset + i];
}

__global__ void kernelUpdateGBestAssign(short *gBestAssign, short *pBestAssign,
                                        int offset, int data_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= data_size)
        return;

    gBestAssign[i] = pBestAssign[offset + i];
}

/*
 * Wrapper to initialize and running PSO on device
 */
extern "C" GBest devicePsoClustering(Data *datas, int *flatDatas, int data_size,
                                     int channel, int particle_size,
                                     int cluster_size, int max_iter)
{
    // Initialize host memory
    int *positions = new int[particle_size * cluster_size * channel];
    int *velocities = new int[particle_size * cluster_size * channel];
    int *pBests = new int[particle_size * cluster_size * channel];
    int *gBest = new int[cluster_size * channel];
    short *posAssign = new short[particle_size * data_size];
    short *pBestAssign = new short[particle_size * data_size];
    short *gBestAssign = new short[data_size];

    // Initialize assignment matrix to cluster 0
    for(int i = 0; i < particle_size * data_size; i++)
    {
        posAssign[i] = 0;
        pBestAssign[i] = 0;

        if(i < data_size)
            gBestAssign[i] = 0;
    }

    initialize(positions, velocities, pBests, gBest, datas, data_size,
               particle_size, cluster_size, channel);

    // Initialize device memory
    int *devPositions, *devVelocities, *devPBests, *devGBest;
    short *devPosAssign, *devPBestAssign, *devGBestAssign;
    int *devDatas;
    float *devFitnesses;

    size_t size = sizeof(int) * particle_size * cluster_size * channel;
    size_t assign_size = sizeof(short) * particle_size * data_size;

    hipMalloc((void**)&devPositions, size);
    hipMalloc((void**)&devVelocities, size);
    hipMalloc((void**)&devPBests, size);
    hipMalloc((void**)&devGBest, sizeof(int) * cluster_size * channel);
    hipMalloc((void**)&devPosAssign, assign_size);
    hipMalloc((void**)&devPBestAssign, assign_size);
    hipMalloc((void**)&devGBestAssign, sizeof(short) * data_size);
    hipMalloc((void**)&devDatas, sizeof(int) * data_size * channel);
    hipMalloc((void**)&devFitnesses, sizeof(float) * particle_size);

    // Copy data from host to device
    hipMemcpy(devPositions, positions, size, hipMemcpyHostToDevice);
    hipMemcpy(devVelocities, velocities, size, hipMemcpyHostToDevice);
    hipMemcpy(devPBests, pBests, size, hipMemcpyHostToDevice);
    hipMemcpy(devGBest, gBest, sizeof(int) * cluster_size * channel,
               hipMemcpyHostToDevice);
    hipMemcpy(devPosAssign, posAssign, assign_size, hipMemcpyHostToDevice);
    hipMemcpy(devPBestAssign, pBestAssign, assign_size, hipMemcpyHostToDevice);
    hipMemcpy(devGBestAssign, gBestAssign, sizeof(short) * data_size,
               hipMemcpyHostToDevice);
    hipMemcpy(devDatas, flatDatas, sizeof(int) * data_size * channel,
               hipMemcpyHostToDevice);

    // Wrap device pointer to device pBests array
    thrust::device_ptr<float> dptrFitness(devFitnesses);
    thrust::device_ptr<float> resPtr;

    // Threads and blocks number
    int threads = 32;
    int blocksPart = (particle_size / threads) + 1;
    int blocksFull = (particle_size * cluster_size * channel / threads) + 1;

    // Iteration
    for (int iter = 0; iter < max_iter; iter++)
    {
        float rp = getRandomClamped();
        float rg = getRandomClamped();

        kernelUpdateParticle<<<blocksFull, threads>>>
            (devPositions, devVelocities, devPBests, devGBest, devPosAssign,
             devDatas, rp, rg, data_size, particle_size, cluster_size, channel);

        kernelUpdatePBest<<<blocksPart, threads>>>
            (devPositions, devPBests, devPosAssign, devPBestAssign,
             devFitnesses, devDatas, data_size, particle_size, cluster_size,
             channel);

        // Get min element
        resPtr = thrust::min_element(dptrFitness, dptrFitness + particle_size);
        // Cast to raw pointer
        int index = resPtr - dptrFitness;

        int offsetParticle = index * cluster_size * channel;
        int offsetAssign = index * data_size;

        // // Update gBest on device
        kernelUpdateGBest<<<((cluster_size * channel) / threads) + 1, threads>>>
            (devGBest, devPBests, offsetParticle, cluster_size, channel);

        // Update gBest assignment matrix on device
        kernelUpdateGBestAssign<<<(data_size / threads) + 1, threads>>>
            (devGBestAssign, devPBestAssign, offsetAssign, data_size);
    }

    // Copy gBest from device to host
    hipMemcpy(gBest, devGBest, sizeof(int) * cluster_size * channel,
               hipMemcpyDeviceToHost);
    hipMemcpy(gBestAssign, devGBestAssign, sizeof(short) * data_size,
               hipMemcpyDeviceToHost);

    // Cleanup
    delete[] positions;
    delete[] velocities;
    delete[] pBests;
    delete[] posAssign;
    delete[] pBestAssign;

    hipFree(devPositions);
    hipFree(devVelocities);
    hipFree(devPBests);
    hipFree(devGBest);
    hipFree(devPosAssign);
    hipFree(devPBestAssign);
    hipFree(devGBestAssign);
    hipFree(devDatas);
    hipFree(devFitnesses);

    GBest gBestReturn;
    gBestReturn.gBestAssign = gBestAssign;
    gBestReturn.arrCentroids = gBest;
    gBestReturn.quantError = devFitness(gBestReturn.gBestAssign, flatDatas,
                                        gBestReturn.arrCentroids, data_size,
                                        cluster_size, channel);

    return gBestReturn;
}
